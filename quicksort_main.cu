#include <stdio.h>
#include <stdlib.h>

#include "support.h"
#include "quicksort_kernel.cu"

int main(int argc, char* argv[]) {
    Timer timer;

    // Initialize host variables ----------------------------------------------
    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *in_h;
    float *in_d;
    unsigned in_elements;
    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    // Input size
    if (argc == 1) {
        in_elements = 10000;
    } else if (argc == 2) {
        in_elements = atoi(argv[1]);
    } else {
        printf("\nInvalid input parameters!"
               "\nUsage: ./quicksort         # Input of size 10000 is used"
               "\nUsage: ./quicksort <m>      # Input of size m is used\n");
        exit(0);
    }
    initVector(&in_h, in_elements);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Input size = %u\n", in_elements);

    // Allocate device variables ----------------------------------------------
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&in_d, in_elements * sizeof(float));
    if (cuda_ret != hipSuccess) FATAL("Unable to allocate device memory: in");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Perform host-side quicksort --------------------------------------------
    printf("Running host-side quick sort...\n"); fflush(stdout);
    startTime(&timer);
    quickSort(in_h, 0, in_elements - 1);
    stopTime(&timer); printf("Host-side quick sort time: %f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(in_d, in_h, in_elements * sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device: in");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel...\n"); fflush(stdout);
    startTime(&timer);

    dim_block.x = BLOCK_SIZE;
    dim_grid.x = 1;

    quicksort_kernel<<<dim_grid, dim_block>>>(in_d, 0, in_elements - 1);
    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) FATAL("Unable to launch kernel");

    stopTime(&timer);
    printf("Device quick sort time: %f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------
    printf("Verifying results...\n"); fflush(stdout);

    float* sorted_h = (float*)malloc(in_elements * sizeof(float));
    hipMemcpy(sorted_h, in_d, in_elements * sizeof(float), hipMemcpyDeviceToHost);

    bool sorted = true;
    for (int i = 1; i < in_elements; i++) {
        if (sorted_h[i] < sorted_h[i - 1]) {
            sorted = false;
            break;
        }
    }

    if (sorted) {
        printf("Sorting successful\n\n");
    } else {
        printf("Sorting failed\n\n");
    }

    // Free memory ------------------------------------------------------------
    hipFree(in_d);
    free(in_h);
    free(sorted_h);

    return 0;
}
