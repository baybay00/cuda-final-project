#include <stdio.h>

#include "support.h"
#include "selectionsort_kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *in_h;
    float *in_d;
    unsigned in_elements;
    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    // Allocate and initialize host memory
    if(argc == 1) {
        in_elements = 100000;
    } else if(argc == 2) {
        in_elements = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./selection_sort         # Input of size 100,000 is used"
           "\n    Usage: ./selection_sort <m>      # Input of size m is used"
           "\n");
        exit(0);
    }
    initVector(&in_h, in_elements);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Input size = %u\n", in_elements);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&in_d, in_elements * sizeof(float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(in_d, in_h, in_elements * sizeof(float),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Timing host-side selection sort ---------------------------------------
    printf("Running host-side selection sort...\n"); fflush(stdout);
    startTime(&timer);

    float* in_h_copy = (float*)malloc(in_elements * sizeof(float));
    memcpy(in_h_copy, in_h, in_elements * sizeof(float));
    bubbleSort(in_h_copy, in_elements); 

    stopTime(&timer); 
    printf("Host-side selection sort time: %f s\n", elapsedTime(timer));

    free(in_h_copy);  // Free the temporary copy

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel...\n"); fflush(stdout);
    startTime(&timer);

    dim_block.x = BLOCK_SIZE;
    dim_grid.x = (in_elements + 2 * BLOCK_SIZE -1)/(2 * BLOCK_SIZE);

    selectionsort_kernel<<<dim_grid, dim_block>>>(in_d, in_elements * sizeof(float));
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    
    
    stopTime(&timer); 
    printf("Device selection sort time: %f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results...\n"); fflush(stdout);

    hipDeviceSynchronize();

    float* sorted_h = (float*)malloc((in_elements) * sizeof(float));
    cuda_ret = hipMemcpy(sorted_h, in_d, in_elements*sizeof(float), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) printf("Unable to copy sorted_h from device to host");

    bool sorted = true;
    for (int i = 1; i < in_elements; i++) {
        if (sorted_h[i] < sorted_h[i - 1]) {
            sorted = false;
            break;
        }
    }

    if(sorted)
    {
        printf("Sorting successful\n\n");
    } else {
        printf("Sorting failed\n\n");
    }
    // Free memory ------------------------------------------------------------

    hipFree(in_d);
    free(in_h);
    free(sorted_h);

    return 0;
}

