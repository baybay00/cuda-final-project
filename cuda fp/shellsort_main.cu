#include <stdio.h>

#include "support.h"
#include "shellsort_kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *in_h;
    float *in_d;
    unsigned in_elements;
    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    // Allocate and initialize host memory
    if(argc == 1) {
        in_elements = 10000;
    } else if(argc == 2) {
        in_elements = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./shell_sort          # Input of size 10,000 is used"
           "\n    Usage: ./shell_sort <m>      # Input of size m is used"
           "\n");
        exit(0);
    }
    initVector(&in_h, in_elements);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Input size = %u\n", in_elements);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&in_d, in_elements * sizeof(float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(in_d, in_h, in_elements * sizeof(float),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Timing host-side shell sort ---------------------------------------
    printf("Running host-side shell sort...\n"); fflush(stdout);
    startTime(&timer);

    float* in_h_copy = (float*)malloc(in_elements * sizeof(float));
    memcpy(in_h_copy, in_h, in_elements * sizeof(float));
    shellSort(in_h_copy, in_elements); 

    stopTime(&timer); 
    printf("Host-side shell sort time: %f s\n", elapsedTime(timer));

    free(in_h_copy);  // Free the temporary copy

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel...\n"); fflush(stdout);
    startTime(&timer);

    dim_block.x = BLOCK_SIZE;
    dim_grid.x = (in_elements + BLOCK_SIZE -1)/(BLOCK_SIZE);

    for(int gap = in_elements/2; gap > 0; gap /= 2)
    {
        shellsort_kernel<<<dim_grid, dim_block, dim_block.x*sizeof(float)>>>(in_d, in_elements, gap);
    }
    // cuda_ret = hipDeviceSynchronize();
    // if(cuda_ret != hipSuccess) printf("unable to launch kernel");
    
    stopTime(&timer); 
    printf("Device shell sort time: %f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results...\n"); fflush(stdout);

    float* sorted_h = (float*)malloc((in_elements) * sizeof(float));
    hipMemcpy(sorted_h, in_d, in_elements*sizeof(float), hipMemcpyDeviceToHost);

    bool sorted = true;
    for (int i = 1; i < in_elements; i++) {
        if (sorted_h[i] < sorted_h[i - 1]) {
            sorted = false;
            break;
        }
    }

    if(sorted)
    {
        printf("Sorting successful\n\n");
    } else {
        printf("Sorting failed\n\n");
    }
    // Free memory ------------------------------------------------------------

    hipFree(in_d);
    free(in_h);
    free(sorted_h);

    return 0;
}

