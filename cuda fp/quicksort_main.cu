#include <stdio.h>
#include "support.h"
#include "quicksort_kernel.cu"

int main(int argc, char* argv[]) {
    Timer timer;

    // Initialize host variables ----------------------------------------------
    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *in_h;
    float *in_d;
    unsigned in_elements;
    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    // Allocate and initialize host memory
    if(argc == 1) {
        in_elements = 100000;
    } else if(argc == 2) {
        in_elements = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./quicksort         # Input of size 100,000 is used"
           "\n    Usage: ./quicksort <m>      # Input of size m is used"
           "\n");
        exit(0);
    }
    initVector(&in_h, in_elements);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Input size = %u\n", in_elements);

    // Allocate device variables ----------------------------------------------
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&in_d, in_elements * sizeof(float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory: in");

    int *d_stack, *d_top;
    cuda_ret = hipMalloc((void**)&d_stack, 2 * in_elements * sizeof(int));  // Stack size can be up to 2*in_elements
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for stack");

    cuda_ret = hipMalloc((void**)&d_top, sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for top");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(in_d, in_h, in_elements * sizeof(float), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device: in");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Timing host-side bubble sort ---------------------------------------
    printf("Running host-side quick sort...\n"); fflush(stdout);
    startTime(&timer);

    quickSort(in_h, 0, in_elements - 1);

    stopTime(&timer); 
    printf("Host-side quick sort time: %f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel...\n"); fflush(stdout);
    startTime(&timer);

    dim_block.x = BLOCK_SIZE;
    dim_grid.x = (in_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;  // Adjust grid size

    quicksort_kernel<<<dim_grid, dim_block>>>(in_d, d_stack, d_top, in_elements);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    
    stopTime(&timer); 
    printf("Device quick sort time: %f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------
    printf("Verifying results...\n"); fflush(stdout);

    float* sorted_h = (float*)malloc((in_elements) * sizeof(float));
    hipMemcpy(sorted_h, in_d, in_elements*sizeof(float), hipMemcpyDeviceToHost);

    bool sorted = true;
    for (int i = 1; i < in_elements; i++) {
        if (sorted_h[i] < sorted_h[i - 1]) {
            sorted = false;
            break;
        }
    }

    if(sorted)
    {
        printf("Sorting successful\n\n");
    } else {
        printf("Sorting failed\n\n");
    }

    // Free memory ------------------------------------------------------------
    hipFree(in_d);
    hipFree(d_stack);
    hipFree(d_top);
    free(in_h);
    free(sorted_h);

    return 0;
}